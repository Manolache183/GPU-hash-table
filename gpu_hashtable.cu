#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
atomicMax
atomicCAS
atomicExch
*/

__device__ int kernelHashFunction(int key, int capacity) {
	unsigned long int hash = key;
	hash = hash * 11 + 7;
	hash = hash % capacity;
	return hash;
}

__global__ void kernelInsertKey(int *keys, int* values, int numKeys, std::pair<int, int>* hashTable, int capacity, int *size) {
	// Compute the global element index this thread should process
  	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i >= numKeys) {
		return;
	}

	int key = keys[i];
	int value = values[i];

	if (key <= 0 || value <= 0) {
		return;
	}

	int hash = kernelHashFunction(key, capacity);
	int j = hash;
	int val;

	while (true) {
		val = atomicCAS(&hashTable[j].first, 0, key);

		if (val == 0) { // Empty slot
			val = atomicExch(&hashTable[j].second, value);
			atomicAdd(size, 1);
			break;
		} else if (val == key) { // Key already exists
			val = atomicExch(&hashTable[j].second, value);
			break;
		}

		// Collision, find next empty slot
		j = (j + 1) % capacity;
	}
}

__global__ void kernelGetKey(int *keys, int *values, int numKeys, std::pair<int, int>* hashTable, int capacity) {
	// Compute the global element index this thread should process
  	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i >= numKeys) {
		return;
	}

	int targetKey = keys[i];

	int hash = kernelHashFunction(targetKey, capacity);

	int j = hash;
	int key;

	while (true) {
		key = hashTable[j].first;

		if (key == 0) { // empty spot
			values[i] = 0;
			return;
		} else if (key == targetKey) { // key found
			values[i] = hashTable[j].second;
			return;
		}

		j = (j + 1) % capacity;
	}
}

__global__ void kernelGetAllPairs(int *keys, int *values, std::pair<int, int>* hashTable, int capacity) {
	// Compute the global element index this thread should process
  	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i >= capacity) {
		return;
	}

	keys[i] = hashTable[i].first;
	values[i] = hashTable[i].second;
}

float loadFactor(int size, int capacity) {
	return ((float) size / (float) capacity);
}

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int initialCapacity) {
	capacity = initialCapacity;
	glbGpuAllocator->_cudaMalloc((void**) &hashTable, capacity * sizeof(std::pair<int, int>));

	hipMemset(hashTable, 0, capacity * sizeof(std::pair<int, int>));

	this->size = 0;
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(hashTable);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int sizeReshape) {
	std::pair<int, int> *newHashTable;

	glbGpuAllocator->_cudaMalloc((void**) &newHashTable, sizeReshape * sizeof(std::pair<int, int>));

	hipMemset(newHashTable, 0, sizeReshape * sizeof(std::pair<int, int>));

	int *device_keys, *device_values, *device_size;
	this->size = 0;

	glbGpuAllocator->_cudaMalloc((void**) &device_keys, this->capacity * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void**) &device_values, this->capacity * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void**) &device_size, sizeof(int));

	hipMemcpy(device_size, &(this->size), sizeof(int), hipMemcpyHostToDevice);

	const int blockSize = 1024;
	size_t block_no = this->capacity / blockSize;
	if (this->capacity % blockSize != 0) {
		block_no++;
	}

	kernelGetAllPairs<<<block_no, blockSize>>>(device_keys, device_values, hashTable, this->capacity);
	hipDeviceSynchronize();

	kernelInsertKey<<<block_no, blockSize>>>(device_keys, device_values, this->capacity, newHashTable, sizeReshape, device_size);
	hipDeviceSynchronize();

	hipMemcpy(&(this->size), device_size, sizeof(int), hipMemcpyDeviceToHost);

	glbGpuAllocator->_cudaFree(hashTable);

	glbGpuAllocator->_cudaFree(device_keys);
	glbGpuAllocator->_cudaFree(device_values);
	glbGpuAllocator->_cudaFree(device_size);

	hashTable = newHashTable;
	this->capacity = sizeReshape;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	if (keys == NULL || values == NULL || numKeys < 0) {
		return false;
	}

	int newSize = this->size + numKeys;

	if (loadFactor(newSize, this->capacity) > 0.8) {
		reshape(3 * newSize / 2);
	}

	int *device_keys, *device_values, *device_size;

	glbGpuAllocator->_cudaMalloc((void**) &device_keys, numKeys * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void**) &device_values, numKeys * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void**) &device_size, sizeof(int));

	hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_size, &(this->size), sizeof(int), hipMemcpyHostToDevice);

	const int blockSize = 1024;
	size_t block_no = numKeys / blockSize;
	if (numKeys % blockSize != 0) {
		block_no++;
	}

	kernelInsertKey<<<block_no, blockSize>>>(device_keys, device_values, numKeys, hashTable, this->capacity, device_size);
	hipDeviceSynchronize();

	hipMemcpy(&(this->size), device_size, sizeof(int), hipMemcpyDeviceToHost);

	glbGpuAllocator->_cudaFree(device_keys);
	glbGpuAllocator->_cudaFree(device_values);
	glbGpuAllocator->_cudaFree(device_size);

	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	if (keys == NULL || numKeys < 0) {
		return NULL;
	}

	int *values, *device_keys, *device_values;

	values = (int*) malloc(numKeys * sizeof(int));
	if (values == NULL) {
		cout << "Error allocating memory for values" << endl;
		exit(1);
	}

	glbGpuAllocator->_cudaMalloc((void**) &device_keys, numKeys * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void**) &device_values, numKeys * sizeof(int));

	hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

	const int blockSize = 1024;
	size_t block_no = numKeys / blockSize;
	if (numKeys % blockSize != 0) {
		block_no++;
	}

	kernelGetKey<<<block_no, blockSize>>>(device_keys, device_values, numKeys, hashTable, capacity);
	hipDeviceSynchronize();

	hipMemcpy(values, device_values, numKeys * sizeof(int), hipMemcpyDeviceToHost);


	glbGpuAllocator->_cudaFree(device_keys);
	glbGpuAllocator->_cudaFree(device_values);

	return values;
}